#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <modes.h>
#include <time.h>
#include "cutil_math.h"
#include "thrust\device_vector.h"
#include "thrust\reduce.h"
#include "thrust\count.h"
#include "thrust\scan.h"
#include "thrust\unique.h"
#include "thrust\remove.h"
#include "thrust\adjacent_difference.h"
#include "TransformMgr.h"
#include "DeformGPU.h"
//#include "StreamDeform.h"


//set it large to avoid flashing in the end of deformation
//if it is too large, the points may not recover when out of target region
#define EPS 1E-9		
#define M_PI       3.14159265358979323846
#define MAX_NUM_GROUP 16

//paramters
float _para;

//window
__device__ __constant__ int WIN_WIDTH;
__device__ __constant__ int WIN_HEIGHT;
int _h_winWidth;
int _h_winHeight;
__device__ __constant__ float MODELVIEW[ 16 ]; 
__device__ __constant__ float INVMODELVIEW[ 16 ];    
__device__ __constant__ float PROJECTION[ 16 ];    
__device__ __constant__ float INVPROJECTION[ 16 ]; 
float _h_modelview[16];
float _h_projection[16];
float *_invModelView;
float *_invProjection;
bool *_deformOn;

thrust::device_vector<hull_type> d_vec_hullSet;

//convex hull
thrust::device_vector<float2> _d_hull;
std::vector<ellipse>* _h_vec_ellipseSet;

//streamline
int _nv;
__device__ __constant__ int NV;

//these two device_ptr are the OpenGL pointer
thrust::device_ptr<float3> _d_ptr_tangent;
thrust::device_ptr<float4> _d_ptr_posClip;
thrust::device_ptr<int> _d_ptr_translucent;

thrust::device_vector<int> d_vec_streamlineLengths;
thrust::device_vector<int> d_vec_streamlineOffsets;
thrust::device_vector<int> d_vec_streamlineLengthsOrig;
thrust::device_vector<int> d_vec_streamlineOffsetsOrig;

thrust::device_vector<float4> _d_vec_pos;
thrust::device_vector<float4> _d_vec_prePos;
thrust::device_vector<float4> _d_vec_origPos;

thrust::device_vector<float2> d_vec_posScreen;

thrust::device_vector<int> _d_vec_lineIndex;
thrust::device_vector<int> _d_vec_lineIndexOrig;
thrust::device_vector<bool> _d_vec_vertexIsFocus;

thrust::device_vector<int> _d_vec_cutPointsMarkOrig;	
thrust::device_vector<int> _d_vec_cutPointsMark;	


thrust::counting_iterator<int> counting_zero(0);
thrust::host_vector<int> *_pickedLineSet;

//lens
float *_lens_center;
//float *_lensDepth_clip;
float4* _lensCenterObject;

DEFORM_MODE *_deformMode;
SOURCE_MODE *_sourceMode;

//StreamDeform* _deformLine;
//
//void SetStreamDeform(void* sd)
//{
//	_deformLine = (StreamDeform*)sd;
//}

__device__ __host__ inline float2 GetXY(float4 pos)
{
	return make_float2(pos.x, pos.y);
}

__device__ inline float3 GetXYZ(float4 v)
{
	return make_float3(v.x, v.y, v.z);
}

__device__ inline float2 rotate(float2 p, float deg)
{
	return make_float2(p.x * cos(deg) - p.y * sin(deg), p.x * sin(deg) + p.y * cos(deg));
}

//Object space-->Camera space-->Clip space-->Screen space
//multiply projection and modelview matrix
__device__ inline float4 Clip2Object(float4 p)//, float modelview[16], float projection[16])
{
	return Clip2Object(p, INVMODELVIEW, INVPROJECTION);
}

__device__ __host__ inline float4 Object2Clip(float4 pos)//, float modelview[16], float projection[16])
{
#ifdef __CUDA_ARCH__ // __CUDA_ARCH__ // 
	return Object2Clip(pos, MODELVIEW, PROJECTION);
#else
	return Object2Clip(pos, _h_modelview, _h_projection);
#endif
}

//only multiply projection matrix
__device__ __host__ inline float4 Camera2Clip(float4 pos)
{
#ifdef __CUDA_ARCH__ 
	return Camera2Clip(pos, PROJECTION);
#else
	return Camera2Clip(pos, _h_projection);
#endif
}

//multiply modelview matrix
__device__ __host__ inline float4 Object2Camera(float4 pos)//, float modelview[16], float projection[16])
{
#ifdef __CUDA_ARCH__ 
	return Object2Camera(pos, MODELVIEW);
#else
	return Object2Camera(pos, _h_modelview);
#endif
}

__device__ __host__ inline float2 Clip2Screen(float2 p)
{
#ifdef __CUDA_ARCH__
	return Clip2Screen(p, WIN_WIDTH, WIN_HEIGHT);
#else
	return Clip2Screen(p, _h_winWidth, _h_winHeight);
#endif
}

__device__ __host__ inline float2 Screen2Clip(float2 p)
{
#ifdef __CUDA_ARCH__
	return Screen2Clip(p, WIN_WIDTH, WIN_HEIGHT);
#else
	return Screen2Clip(p, _h_winWidth, _h_winHeight);
#endif
}

__device__ __host__ inline float2 Object2Screen(float4 p)
{
	return Clip2Screen(GetXY(Object2Clip(p)));
}

//__device__ float2 force_linear(float2 distance, float c)
//{
//	float2 force = c * distance;
//	return force;
//}

__device__ inline float3 GetNormal(int i, int *lineIndex, float4* pos)
{
	float4 A, B, C;
	float4 pos_camera = Object2Camera(pos[i]);
	if(i >= (NV - 2) || lineIndex[i] != lineIndex[i + 1])	//last point of a streamline
	{
		A = Object2Camera(pos[i - 2]);
		//B = Object2Camera(pos[i - 1]);
		C = pos_camera;
	}
	else if(i <= 1 || lineIndex[i] != lineIndex[i - 1])	//first point of a streamline
	{
		A = pos_camera;
		//B = Object2Camera(pos[i + 1]);
		C = Object2Camera(pos[i + 2]);
	}
	else
	{
		A = Object2Camera(pos[i - 1]);
		//B = pos_camera;
		C = Object2Camera(pos[i + 1]);
	}
//	float3 BC = GetXYZ(C - B);
//	float3 AB = make_float3(B - A);
//	float3 binormal = cross(AB, BC);//AB X BC
	return normalize(GetXYZ(C - A));// crossProduct(AB, binormal);
}

struct functor_computeNormal
{
	int* lineIndex;
	float4* pos;
	template <typename T>
	__device__ void operator() (T t)
	{
		thrust::get<1>(t) = GetNormal(thrust::get<0>(t), lineIndex, pos);//make_float3(0,1,0);//
	}

	functor_computeNormal(int *_lineIndex, float4* _pos)
	{
		lineIndex = _lineIndex;
		pos = _pos;
	}
};

__device__ inline float4 Recover3DPosition(float4 pos, float4 origPos, float moveSpeed)
{
	float4 dir = origPos - pos;
	float4 temp;
	if(length(dir) > 0.1)
	{
		float4 moveStep =  5 * moveSpeed * dir;
		temp = pos + moveStep;
	}
	else
		temp = origPos;

	return temp;
}

__device__ inline float2 CatmullRom_Evaluate(float2 p0, float2 p1, float2 p2, float2 p3, float t)
{
	const float c0 = ((-t + 2.0f) * t - 1.0f) * t * 0.5f;
	const float c1 = (((3.0f * t - 5.0f) * t) * t + 2.0f) * 0.5f;
	const float c2 = ((-3.0f * t + 4.0f) * t + 1.0f) * t * 0.5f;
	const float c3 = ((t - 1.0f) * t * t) * 0.5f;
	
	float2 ret = c0 * p0 + c1 * p1 + c2 * p2 + c3 * p3;
	return ret;
}

__device__ float2 ProjectPoint2Line(float2 p, float2 p1, float2 p2)
{
	float2 v = p - p1;
	float2 v2 = p2 - p1;
	return p1 + dot(v, v2) / dot(v2, v2) * v2;
}

//http://geomalgorithms.com/a02-_lines.html
//Distance of a Point to an finite Line
__device__ inline float Point2Line(float2 p, float2 p0, float2 p1)
{
	float ret;
	float m = p0.y - p1.y;
	float n = p1.x - p0.x;
	float a = m * p.x + n * p.y + (p0.x * p1.y - p1.x * p0.y);
	float b = sqrtf(m * m + n * n);
	if(dot(p - p0, p1 - p0) < 0)
		ret = length(p - p0);
	else if( dot(p - p1, p0 - p1) < 0)
		ret = length(p - p1);
	else
		ret = abs(a / b);
	return ret;
}


__device__ inline float Vector2Angle(float2 v)
{
	float t = atan2(v.y, v.x);
	if(t < 0)
		t += 2 * M_PI; 
	return t;
}

__device__ inline float radius_ellipse(ellipse e, float2 dir2Center)
{
	//Polar form relative to center: http://en.wikipedia.org/wiki/Ellipse
	float t = Vector2Angle(dir2Center);

	//rotate to canonical position
	t = t - e.angle;	
	return e.a * e.b / sqrt(pow(e.b * cos(t),2) + pow(e.a * sin(t),2));
}

__device__ inline float radius_blade(ellipse e, float2 center)
{
	float parallelDist = 0.85 * length(center - make_float2(e.x, e.y));
	return 1.25 * e.b * ( tanh(- 8.0 * parallelDist / e.a + 6.0) + 1) * 0.5;
}

__device__ inline bool inOneEllipse(ellipse e, float2 p, float radius)
{
	float2 center = make_float2(e.x, e.y);
	float2 dir2Center = p - center;
	return length(dir2Center) < radius;
}

__device__ inline bool inOneEllipse(ellipse e, float2 p)
{
	float2 center = make_float2(e.x, e.y);
	float2 dir2Center = p - center;
	float radius = radius_ellipse(e, dir2Center);
	return length(dir2Center) < radius;
}

__device__ inline bool inAnyGroup(ellipse* ellipseSet, float* radiusOuter, int cnt, float4 p)
{
	for(int i = 0; i < cnt; i++)
		if(inOneEllipse(ellipseSet[i], Object2Screen(p), radiusOuter[i]))//length(make_float2(ellipseSet[i].x,ellipseSet[i].y) - Object2Screen(p)) < radiusOuter[i])
			return true;
	return false;
}

__device__ inline bool inOneBlade(ellipse e, float radius, float2 p)
{
	float2 line[2];
	line[0] = make_float2(e.x - e.a * cos(e.angle), e.y - e.a * sin(e.angle));
	line[1] = make_float2(e.x + e.a * cos(e.angle), e.y + e.a * sin(e.angle));
	float2 center = ProjectPoint2Line(p, line[0], line[1]);
	return length(center - p) < radius;
}

__device__ inline bool inOneBlade(ellipse e, float2 p)
{
	float2 line[2];
	line[0] = make_float2(e.x - e.a * cos(e.angle), e.y - e.a * sin(e.angle));
	line[1] = make_float2(e.x + e.a * cos(e.angle), e.y + e.a * sin(e.angle));
	float2 center = ProjectPoint2Line(p, line[0], line[1]);
	float radius = radius_blade(e, center);
	return length(center - p) < radius;
}

__device__ inline bool inAnyGroupLine(ellipse* ellipseSet, float* radiusOuter, int cnt, float4 p)
{
	for(int i = 0; i < cnt; i++)
	{
		ellipse e = ellipseSet[i];
		//float2 line[2];
		//line[0] = make_float2(e.x - e.a * cos(e.angle), e.y - e.a * sin(e.angle));
		//line[1] = make_float2(e.x + e.a * cos(e.angle), e.y + e.a * sin(e.angle));
		//float2 v_screen = Object2Screen(p);
		//float2 center = ProjectPoint2Line(v_screen, line[0], line[1]);
		if(inOneBlade(e, radiusOuter[i], Object2Screen(p)))
			return true;
	}
	return false;
}

__device__ inline bool IsBetweenAngles(float v, float a, float b)
{
	float a_b = 0;
	if((a - b) > M_PI)
		return (v >= a || v < b);
	else if((b - a) > M_PI)
		return (v >= b || v < a);
	else
		return (v >= a && v < b) || (v >= b && v < a);
}

__device__ inline float CatmullRomAngle(float a0, float2 center,
	float2 p0, float2 p1, float2 p2, float2 p3)
{
	const int ndivs = 16;
	float angles[ndivs];
	float a_diff_min = FLT_MAX;
	float a_closest = 0;
	float2 iP_closest;
	for(int i = 0; i < ndivs; i++)
	{
		float t = (float)(i) / (float)(ndivs- 1);
		float2 iP = CatmullRom_Evaluate(p0, p1, p2, p3, t);
		float a = Vector2Angle(iP - center);
		if(abs(a - a0) < a_diff_min)
		{
			a_diff_min = abs(a - a0);
			iP_closest = iP;
		}
	}
	return length(iP_closest - center);
}

__device__ inline float radius_hull(hull_type hull, float2 hullCenter, float2 p)
{
	//float2 closest;
	//GetClosestHullPointCatmullRom(closest, p, hull.v, hull.nv);
	float ret = 0;
	float t0 = Vector2Angle(p - hullCenter);
	float hullAngles[HULL_SIZE];
	for(int i = 0; i < hull.nv; i++)
	{
		hullAngles[i] = Vector2Angle(hull.v[i] - hullCenter);
	}
	for(int i = 0; i < hull.nv;i++)
	{
		if(IsBetweenAngles(t0, hullAngles[i], hullAngles[(i + 1) % hull.nv]))
		{
			ret = CatmullRomAngle(t0, hullCenter, 
				hull.v[(i - 1 + hull.nv)% hull.nv], hull.v[i], 
				hull.v[(i + 1) % hull.nv], hull.v[(i + 2) % hull.nv]);
			//ret = length(hull.v[i] - hullCenter);
			break;
		}
	}
	return ret;
	//return length(closest - hullCenter);
}

__device__ inline float G(float x, float r)
{
	return pow((r - 1), 2) / ( - r * r * x + r) + 2 - 1 / r;
}

//kernels
__global__ void kernel_convex(float4* pos, float4* pos_clip, float2* pos_screen,
	float4* prevPos, float4* origPos, int *lineIndex, bool *vertexIsFocus, int _nv, 
	ellipse* ellipseSet, hull_type* hullSet, int nEllipse, 
	DEFORM_MODE deformMode, float pa)//, unsigned int nv)
{
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= _nv)		//if more than the number of vertices
		return;

	float2 v_screen = pos_screen[i];
	float2 orig_screen= Object2Screen(origPos[i]);
	float2 forceAll= make_float2(0, 0);
	float moveSpeed = 0.01;
	const float transRatio = 0.2;
	float generalSize = (ellipseSet[0].a + ellipseSet[0].b) * 0.5;

	float radiusOuterAll[MAX_NUM_GROUP]; //including the focus and transition region

	bool inAnyEllipse = false;
	for(int ie = 0; ie < nEllipse; ie++)
	{
		float2 center;
		ellipse e = ellipseSet[ie];
		if(DEFORM_MODE::MODE_LINE == deformMode)
		{
			float2 line[2];
			line[0] = make_float2(e.x - e.a * cos(e.angle), e.y - e.a * sin(e.angle));
			line[1] = make_float2(e.x + e.a * cos(e.angle), e.y + e.a * sin(e.angle));
			center = ProjectPoint2Line(v_screen, line[0], line[1]);
		}
		else
			center = make_float2(e.x, e.y);
		float2 dir2Center = v_screen - center;	//current distance to the center

		float radius;

		if(DEFORM_MODE::MODE_HULL == deformMode)
			radius = 1.2 * radius_hull(hullSet[ie], center, v_screen);
		else if(DEFORM_MODE::MODE_LINE == deformMode)
		{
			//float parallelDist = 0.85 * length(center - make_float2(e.x, e.y));
			//radius = 1.25 * e.b * ( tanh(- 8.0 * parallelDist / e.a + 6.0) + 1) * 0.5;
			radius = radius_blade(e, center);
		}
		else//		if(DEFORM_MODE::MODE_ELLIPSE == deformMode)
			radius = 1.2 * radius_ellipse(e, dir2Center);
		//float transWidth = generalSize * transRatio; //size of transition region, _d_ / _r_
		float r = 0.5;
		radiusOuterAll[ie] = radius / r;// (radius + transWidth) ;
		
		float origDist2Center = length(orig_screen - center);	//original distance to the center
		if(origDist2Center <= radiusOuterAll[ie] && (false == vertexIsFocus[i]))
		{
			inAnyEllipse = true;
			//force from neighboring vertices
			if(deformMode == DEFORM_MODE::MODE_ELLIPSE)
			{
				if(i != 0 && i != (_nv - 1))
				{
					if((lineIndex[i] == lineIndex[i + 1]) && (lineIndex[i] == lineIndex[i - 1])) //if it is not the first vertex or last vertex on the line
					{
						float2 pre_v1_screen = Object2Screen(prevPos[i + 1]);//projection * modelview * v;
						float2 pre_v_1_screen = Object2Screen(prevPos[i - 1]);//projection * modelview * v;
						float2 edge_v1 = pre_v1_screen - v_screen;
						float2 edge_v_1 = pre_v_1_screen - v_screen;// mul(10,minus(pre_v_1_screen, v));
						float edgeLength_v1 = length(edge_v1);
						float edgeLength_v_1 = length(edge_v_1);
						// if the parameter is too small, the lines are jaggy
						// if the parameter is too large, the lines tend to be straight
						if(edgeLength_v1 > (edgeLength_v_1 * 1.001) )
							forceAll += edge_v1 * 2;	
						else if(edgeLength_v_1 > (edgeLength_v1 * 1.001) )
							forceAll += edge_v_1 * 2;
					}
				}
			}
		
			float2 dir = normalize(dir2Center);

			float desiDist2Center = G(origDist2Center / radiusOuterAll[ie], r) * radiusOuterAll[ie];//radius + origDist2Center / (radius / transWidth + 1.0);		//distance to the center for the destination position
			float dist2Center = length(dir2Center);	//current distance to the center 
			float dist2Desire = desiDist2Center - dist2Center;
			forceAll += (dir * dist2Desire);
		}
	}
	if(length(forceAll) >= generalSize * 0.1)
	{
		v_screen += moveSpeed * forceAll;

		float2 v = Screen2Clip(v_screen);
		pos_clip[i].x = v.x;//ELLIPSE_CENTER.x / WIN_WIDTH * 2.0 - 1.0;//v.x;
		pos_clip[i].y = v.y;//ELLIPSE_CENTER.y / WIN_HEIGHT * 2.0 - 1.0;//v.y;
		pos[i] = Clip2Object(pos_clip[i]);
	}
	if(	!inAnyEllipse|| vertexIsFocus[i])
	{
		//make sure it would not recover into the deformation region
		pos[i] = Recover3DPosition(pos[i], origPos[i], moveSpeed);;
		pos_clip[i] = Object2Clip(pos[i]);
	}
}

void RestorePos()
{
	thrust::copy(_d_vec_origPos.begin(), _d_vec_origPos.end(), _d_vec_pos.begin());
}

void RestoreConnectivity()
{
	d_vec_streamlineLengths = d_vec_streamlineLengthsOrig;
	d_vec_streamlineOffsets = d_vec_streamlineOffsetsOrig;
	_d_vec_cutPointsMark = _d_vec_cutPointsMarkOrig;
}

struct functor_Object2Clip
{
	__device__ float4 operator() (float4 p)
	{
		return Object2Clip(p);
	}
};

struct functor_Object2Screen
{
	__device__ float2 operator() (float4 p)
	{
		return Object2Screen(p);
	}
};


struct functor_Clip2Screen
{
	__device__ float2 operator() (float4 p)
	{
		return Clip2Screen(GetXY(p));
	}
};

struct functor_UpdateVertexIsFocusByLens
{
	bool* vertexIsFocus;
	float4* posClip;
	float2* posScreen;
	ellipse e;
	float lens_z_clip;
	template <typename T>
	__device__ void operator() (T t)
	{
		int offset = thrust::get<0>(t);
		int len = thrust::get<1>(t);
		bool passed = false;
		
		for(int i = 0; i < len; i++)
		{
			float2 p_screen = posScreen[offset + i];
			float p_depth = posClip[offset + i].z;
			float2 dirFromCenter = p_screen - make_float2(e.x, e.y);
			float radius = radius_ellipse(e, dirFromCenter);
			if(length(dirFromCenter) < radius && p_depth < lens_z_clip)
				passed = true;
		}
		for(int i = 0; i < len; i++)
			vertexIsFocus[offset + i] = !passed;
	}

	functor_UpdateVertexIsFocusByLens(bool* _vertexIsFocus, float4* _posClip, float2* _posScreen, 
		ellipse _e, float _lens_z_clip)
	{
		vertexIsFocus = _vertexIsFocus;
		posClip = _posClip;
		posScreen = _posScreen;
		e = _e;
		lens_z_clip = _lens_z_clip;
	}
};

void ResetVertexIsFocus()
{
	_d_vec_vertexIsFocus.assign(_nv, false);
}

void UpdateVertexIsFocusByLens()
{
	float4 lensCenterClip = Object2Clip(*_lensCenterObject);
	float lensDepth_clip = lensCenterClip.z;

	thrust::device_vector<float2> d_vec_origPosScreen(_nv);
	thrust::device_vector<float4> d_vec_origPosClip(_nv);
	thrust::transform(_d_vec_origPos.begin(), _d_vec_origPos.end(), d_vec_origPosClip.begin(), functor_Object2Clip());
	thrust::transform(d_vec_origPosClip.begin(), d_vec_origPosClip.end(), d_vec_origPosScreen.begin(), functor_Clip2Screen());

	////http://stackoverflow.com/questions/3717226/radius-of-projected-sphere
	//use the original position to solve the vibrating problem, because when deformed streamline changes depth
	if(_h_vec_ellipseSet->size() > 0)
	{
		ellipse ell = _h_vec_ellipseSet->front();
		thrust::for_each(
			thrust::make_zip_iterator(thrust::make_tuple(d_vec_streamlineOffsetsOrig.begin(), d_vec_streamlineLengthsOrig.begin(), counting_zero)),
			thrust::make_zip_iterator(thrust::make_tuple(d_vec_streamlineOffsetsOrig.end(), d_vec_streamlineLengthsOrig.end(), counting_zero + d_vec_streamlineOffsets.size())),
			functor_UpdateVertexIsFocusByLens(
				thrust::raw_pointer_cast(_d_vec_vertexIsFocus.data()),
				thrust::raw_pointer_cast(d_vec_origPosClip.data()), 
				thrust::raw_pointer_cast(d_vec_origPosScreen.data()),
				ell,
				lensDepth_clip));
	}
}

struct functor_GetLineIndexInRange 
{
	float min[3];
	float max[3];
	template <typename T>
	__device__ void operator() (T t)
	{
		int idx = thrust::get<0>(t);
		float4 p = thrust::get<1>(t);
		if(p.x >= min[0] && p.x < max[0]
		&&	p.y >= min[1] && p.y < max[1]
		&&	p.z >= min[2] && p.z < max[2])
			thrust::get<2>(t) = idx;
		else
			thrust::get<2>(t) = -1;
	}

	functor_GetLineIndexInRange(float _min[3], float _max[3])
	{
		for(int i = 0; i < 3; i++)
		{
			min[i] = _min[i];
			max[i] = _max[i];
		}
	}
};

thrust::host_vector<int> PickStreamByBlockCUDA(float min[3], float max[3])
{
	vector<int> picked;
	thrust::device_vector<int> result(_nv, -1);
	thrust::for_each(
		thrust::make_zip_iterator(thrust::make_tuple(
		_d_vec_lineIndexOrig.begin(), _d_vec_origPos.begin(), result.begin())), 
		thrust::make_zip_iterator(thrust::make_tuple(
		_d_vec_lineIndexOrig.end(),	_d_vec_origPos.end(), result.end())), 
		functor_GetLineIndexInRange(min, max));

	thrust::device_vector<int>::iterator newEnd = thrust::unique(result.begin(), result.end());
	newEnd = thrust::remove(result.begin(), newEnd, -1);
	thrust::host_vector<int> h_result(result.begin(), newEnd);
	return h_result;
}

struct functor_GenLineIndex
{
	int* index;

	template <typename T>
	__device__ void operator() (T t)
	{
		int offset = thrust::get<0>(t);
		int length = thrust::get<1>(t);
		int lineIdx = thrust::get<2>(t);
		
		for(int i = 0; i < length; i++)
		{
			index[offset + i] = lineIdx;
		}
	}

	functor_GenLineIndex(int* _index)
	{
		index = _index;
	}
};

// Given three colinear points p, q, r, the function checks if
// point q lies on line segment 'pr'
__device__ inline bool onSegment(float2 p, float2 q, float2 r)
{
    if (q.x <= max(p.x, r.x) && q.x >= min(p.x, r.x) &&
        q.y <= max(p.y, r.y) && q.y >= min(p.y, r.y))
       return true;
 
    return false;
}

//*******from http://www.geeksforgeeks.org/check-if-two-given-line-segments-intersect/
// To find orientation of ordered triplet (p, q, r).
// The function returns following values
// 0 --> p, q and r are colinear
// 1 --> Clockwise
// 2 --> Counterclockwise
__device__  inline int orientation(float2 p, float2 q, float2 r)
{
    // See 10th slides from following link for derivation of the formula
    // http://www.dcs.gla.ac.uk/~pat/52233/slides/Geometry1x1.pdf
    int val = (q.y - p.y) * (r.x - q.x) -
              (q.x - p.x) * (r.y - q.y);
 
    if (val == 0) return 0;  // colinear
 
    return (val > 0)? 1: 2; // clock or counterclock wise
}
 
// The main function that returns true if line segment 'p1q1'
// and 'p2q2' intersect.
__device__  inline bool doIntersect(float2 p1, float2 q1, float2 p2, float2 q2)
{
    // Find the four orientations needed for general and
    // special cases
    int o1 = orientation(p1, q1, p2);
    int o2 = orientation(p1, q1, q2);
    int o3 = orientation(p2, q2, p1);
    int o4 = orientation(p2, q2, q1);
 
    // General case
    if (o1 != o2 && o3 != o4)
        return true;
 
    // Special Cases
    // p1, q1 and p2 are colinear and p2 lies on segment p1q1
    if (o1 == 0 && onSegment(p1, p2, q1)) return true;
 
    // p1, q1 and p2 are colinear and q2 lies on segment p1q1
    if (o2 == 0 && onSegment(p1, q2, q1)) return true;
 
    // p2, q2 and p1 are colinear and p1 lies on segment p2q2
    if (o3 == 0 && onSegment(p2, p1, q2)) return true;
 
     // p2, q2 and q1 are colinear and q1 lies on segment p2q2
    if (o4 == 0 && onSegment(p2, q1, q2)) return true;
 
    return false; // Doesn't fall in any of the above cases
}

	//thrust::for_each(
	//	thrust::make_zip_iterator(thrust::make_tuple(_d_vec_cutPointsMark.begin(),counting_zero)),
	//	thrust::make_zip_iterator(thrust::make_tuple(_d_vec_cutPointsMark.end(), counting_zero+ d_vec_filledOffset.size())),
	//	functor_ComputeCutPointsWithLine(
	//		thrust::raw_pointer_cast(d_vec_ellipseSet.data()),
	//		_h_vec_ellipseSet->size(),
	//		thrust::raw_pointer_cast(d_vec_posScreen.data())
	//		));

//__device__ __host__ inline bool InsideBlade(float2 p, ellipse e)
//{
//
//}

struct functor_ComputeCutPointsWithLine
{
	ellipse* ellipseSet;
	int numEllipses;
	float2* posScreen;

	template <typename T>
	__device__ void operator() (T t)
	{
		bool vertexIsFocus = thrust::get<1>(t);
		int cutPointsMark = thrust::get<0>(t);
		if(vertexIsFocus)
		{
			if(2 == cutPointsMark)
				thrust::get<0>(t) = 0;
			return;
		}

		int index = thrust::get<2>(t);
		if(0 == index )//when index == 0, posScreen[index - 1] has no value
			return;

		//meaning of the values of cutPointMark:
		//0: not any cut point
		//1: the first vertex of a streamline
		//2: the cut point
		if(2 == cutPointsMark)		//see weather make it 0
		{
			//no use!!!
			//recover the cut points that have left the ellipse region
			//bool insideAnyBlades = false;
			//for(int j = 0; j < numEllipses; j++)
			//{
			//	if(inOneBlade(ellipseSet[j], ellipseSet[j].b * 0.8, posScreen[index]))
			//		insideAnyBlades = true;
			//}
			//if(!insideAnyBlades)
			//	thrust::get<0>(t) = 0;
		}	
		else if(0 == cutPointsMark) {
			for(int j = 0; j < numEllipses; j++)
			{
				ellipse e = ellipseSet[j];
				float2 line[2];
				line[0] = make_float2(e.x - e.a * cos(e.angle), e.y - e.a * sin(e.angle));
				line[1] = make_float2(e.x + e.a * cos(e.angle), e.y + e.a * sin(e.angle));
				if(doIntersect(
					posScreen[index - 1], posScreen[index], 
					line[0], line[1]))
					thrust::get<0>(t) = 2;
			}
		}
		else //(1 == cutPointsMark )	
		{
			//doing nothing
		}
	}

	functor_ComputeCutPointsWithLine(
		ellipse* _ellipseSet, int _numEllipses, float2* _posScreen)
	{
		ellipseSet = _ellipseSet;
		numEllipses = _numEllipses;
		posScreen = _posScreen;
	}
};
	//thrust::for_each(
	//	thrust::make_zip_iterator(thrust::make_tuple(_d_vec_cutPointsMark.begin(), d_vec_filledOffset.begin(), counting_zero)),
	//	thrust::make_zip_iterator(thrust::make_tuple(_d_vec_cutPointsMark.end(), d_vec_filledOffset.end(), counting_zero + _d_vec_cutPointsMark.size())),
	//	functor_AssignFilledOffsetsFramMask()
	//	);
struct functor_AssignFilledOffsetsFramMask
{
	template <typename T>
	__device__ void operator() (T t)
	{
		int cutPointsMask = thrust::get<0>(t);
		int idx = thrust::get<2>(t);
		if(1 == cutPointsMask || 2 == cutPointsMask)
			thrust::get<1>(t) = idx;
	}
};

struct functor_UpdateCutPointsMarkByConnectivity
{
	int *cutPointsMark;

	//template <typename T>
	__device__ void operator() (int offset)
	{
//		int offset = thrust::get<0>(t);
		cutPointsMark[offset] = 1;
	}

	functor_UpdateCutPointsMarkByConnectivity(int *_cutPointsMark)
	{
		cutPointsMark = _cutPointsMark;
	}
};


struct functor_PushFromLine
{
	ellipse *ellipseSet;
	int numEllipses;

	template <typename T>
	__device__ void operator() (T t)
	{
		if(thrust::get<1>(t) < 0)
			return;
		float2 p_screen = thrust::get<0>(t);
		for(int j = 0; j < numEllipses; j++)
		{
			ellipse e = ellipseSet[j];
			float2 line[2];
			line[0] = make_float2(e.x - e.a * cos(e.angle), e.y - e.a * sin(e.angle));
			line[1] = make_float2(e.x + e.a * cos(e.angle), e.y + e.a * sin(e.angle));
			if(Point2Line(p_screen, line[0], line[1]) < 2)
			{
				float2 p_proj = ProjectPoint2Line(p_screen, line[0], line[1]);
				float2 dir = normalize(p_screen - p_proj);
				p_screen = p_screen + dir * 4;
			}
		}
		thrust::get<0>(t) = p_screen;
	}

	functor_PushFromLine(ellipse* _ellipseSet, int _numEllipses)
	{
		ellipseSet = _ellipseSet;
		numEllipses = _numEllipses;
	}
};

struct functor_Screen2ObjectOnSamePlane
{
	template <typename T>
	__device__ void operator() (T t)
	{
		float4 pos_clip = thrust::get<1>(t);
		float2 v_screen = thrust::get<2>(t);

		float2 v = Screen2Clip(v_screen);
		pos_clip.x = v.x;//ELLIPSE_CENTER.x / WIN_WIDTH * 2.0 - 1.0;//v.x;
		pos_clip.y = v.y;//ELLIPSE_CENTER.y / WIN_HEIGHT * 2.0 - 1.0;//v.y;
		thrust::get<0>(t) = Clip2Object(pos_clip);
	}
};

struct functor_UpdateLineIndexWithPickedLine
{
	int* offsets;
	int* lengths;
	bool* vertexIsFocus;

//	template <typename T>
	__device__ void operator() (int picked)
	{
		int offset = offsets[picked];
		int length = lengths[picked];
		for(int i = 0; i < length; i++)	{
			int idx = offset + i;
			vertexIsFocus[idx] = true;
		}
	}

	functor_UpdateLineIndexWithPickedLine(int* _offsets, int* _lengths, bool* _vertexIsFocus)
	{
		offsets = _offsets;
		lengths = _lengths;
		vertexIsFocus = _vertexIsFocus;
	}
};

void UpdateLineIndex()
{
	thrust::for_each(
		thrust::make_zip_iterator(thrust::make_tuple(d_vec_streamlineOffsets.begin(), d_vec_streamlineLengths.begin(), counting_zero)),
		thrust::make_zip_iterator(thrust::make_tuple(d_vec_streamlineOffsets.end(), d_vec_streamlineLengths.end(), counting_zero + d_vec_streamlineOffsets.size())),
		functor_GenLineIndex(thrust::raw_pointer_cast(_d_vec_lineIndex.data())));
}

void ComputeCutPoints()
{
	//this offset use the index to mark the first element of a streamline, 
	//and use -1 to mark the others
	thrust::device_vector<int> d_vec_filledOffset;	
	d_vec_filledOffset.assign(_nv, -1);
	//thrust::for_each(
	//	//d_vec_streamlineOffsets.begin(), d_vec_streamlineOffsets.end(),
	//	thrust::make_zip_iterator(thrust::make_tuple(d_vec_streamlineOffsets.begin(), d_vec_streamlineLengths.begin())),
	//	thrust::make_zip_iterator(thrust::make_tuple(d_vec_streamlineOffsets.end(), d_vec_streamlineLengths.end())),
	//functor_AssignFilledOffsets(
	//	thrust::raw_pointer_cast(d_vec_filledOffset.data()),
	//	thrust::raw_pointer_cast(_d_vec_vertexIsFocus.data())
	//	));

	thrust::device_vector<float2> d_vec_posScreen(_nv);
	thrust::device_vector<float4> d_vec_posClip(_nv);
	thrust::transform(_d_vec_pos.begin(), _d_vec_pos.end(), d_vec_posClip.begin(), functor_Object2Clip());
	thrust::transform(d_vec_posClip.begin(), d_vec_posClip.end(), d_vec_posScreen.begin(), functor_Clip2Screen());

	thrust::device_vector<ellipse> d_vec_ellipseSet = *_h_vec_ellipseSet;

	//compute the new _d_vec_cutPointsMark
	thrust::for_each(
		thrust::make_zip_iterator(thrust::make_tuple(_d_vec_cutPointsMark.begin(), _d_vec_vertexIsFocus.begin(), counting_zero)),
		thrust::make_zip_iterator(thrust::make_tuple(_d_vec_cutPointsMark.end(), _d_vec_vertexIsFocus.end(), counting_zero+ d_vec_filledOffset.size())),
		functor_ComputeCutPointsWithLine(
			thrust::raw_pointer_cast(d_vec_ellipseSet.data()),
			_h_vec_ellipseSet->size(),
			thrust::raw_pointer_cast(d_vec_posScreen.data())
			));

	//use _d_vec_cutPointsMark to update d_vec_filledOffset
	thrust::for_each(
		thrust::make_zip_iterator(thrust::make_tuple(_d_vec_cutPointsMark.begin(), d_vec_filledOffset.begin(), counting_zero)),
		thrust::make_zip_iterator(thrust::make_tuple(_d_vec_cutPointsMark.end(), d_vec_filledOffset.end(), counting_zero + _d_vec_cutPointsMark.size())),
		functor_AssignFilledOffsetsFramMask()
		);

	//use d_vec_filledOffset to update connectivity
	thrust::device_vector<int>::iterator filledOffsetEnd = thrust::remove(d_vec_filledOffset.begin(), d_vec_filledOffset.end(), -1); 
	int newSize = filledOffsetEnd - d_vec_filledOffset.begin();
	d_vec_streamlineOffsets.resize(newSize);
	thrust::copy(d_vec_filledOffset.begin(), filledOffsetEnd, d_vec_streamlineOffsets.begin());
	thrust::device_vector<int> d_vec_streamlineLengthsTmp(newSize);
	thrust::adjacent_difference(d_vec_streamlineOffsets.begin(), d_vec_streamlineOffsets.end(), d_vec_streamlineLengthsTmp.begin());
	d_vec_streamlineLengths.resize(newSize);
	thrust::copy(d_vec_streamlineLengthsTmp.begin() + 1, d_vec_streamlineLengthsTmp.end(), d_vec_streamlineLengths.begin());
	d_vec_streamlineLengths.back() = _nv - d_vec_streamlineOffsets.back();
	
	UpdateLineIndex();
}

void UpdateLineIndexWithPickedLine()
{
	//thrust::for_each(
	//	thrust::make_zip_iterator(thrust::make_tuple(
	//		d_vec_streamlineOffsets.begin(), d_vec_streamlineLengths.begin(), counting_zero)),
	//	thrust::make_zip_iterator(thrust::make_tuple(
	//		d_vec_streamlineOffsets.end(), d_vec_streamlineLengths.end(), counting_zero + d_vec_streamlineOffsets.size())),
	//	functor_GenLineIndex(thrust::raw_pointer_cast(_d_vec_lineIndex.data()))
	//	);

	//cout<<"cnt2:"<<_d_vec_lineIndex.back()<<endl;

	//cout<<"*****"<<endl;
	//_d_vec_lineIndex = _d_vec_lineIndexOrig;
	thrust::device_vector<int> picked_lineSet;// = *_pickedLineSet;
	picked_lineSet.assign(_pickedLineSet->begin(), _pickedLineSet->end());

	thrust::for_each(picked_lineSet.begin(), picked_lineSet.end(),
		functor_UpdateLineIndexWithPickedLine(
			thrust::raw_pointer_cast(d_vec_streamlineOffsetsOrig.data()),
			thrust::raw_pointer_cast(d_vec_streamlineLengthsOrig.data()),
			thrust::raw_pointer_cast(_d_vec_vertexIsFocus.data()))
		);
	//for(int i = 0; i < _d_vec_lineIndex.size(); i+= 100)	{
	//	cout<<_d_vec_lineIndex[i] << ", ";
	//}
	//then reverse the sign of the vertices of the picked streamlines

}


//n is the number of vertices
void SetVertexCoords(float* data, int n)
{
	_nv = n;
	_d_vec_pos.assign((float4*)data, (float4*)data + _nv); 
	
	hipMemcpyToSymbol(HIP_SYMBOL(NV), &_nv, sizeof(int));
	d_vec_posScreen.resize(_nv);
	_d_vec_origPos.resize(_nv);
	_d_vec_lineIndexOrig.resize(_nv);
	_d_vec_lineIndex.resize(_nv);
	_d_vec_vertexIsFocus.resize(_nv);

	_d_vec_cutPointsMark.resize(_nv, 0);	//use 0 to mark the vertex that are not cut point	
	_d_vec_cutPointsMarkOrig.resize(_nv, 0);	//use 0 to mark the vertex that are not cut point	


	//_d_vec_IsCutPoint.resize(_nv);
	//_d_vec_IsCutPoint.assign(_nv, false);
	
	//make a copy
	thrust::copy(_d_vec_pos.begin(), _d_vec_pos.end(), _d_vec_origPos.begin());
}

void SetLens(VECTOR4* lensCenterObject)
{
	_lensCenterObject = (float4*)lensCenterObject;
}

void SetMode(DEFORM_MODE *deformMode, SOURCE_MODE *sourceMode)
{
	_deformMode = deformMode;
	_sourceMode = sourceMode;
}

void SetHull(std::vector<hull_type> *hullSet)
{
	if(hullSet->size() > MAX_NUM_GROUP)
	{
		cout<<"exceeded the maximum number of groups..."<<endl;
		exit(1);
	}
	//_hullSet = hullSet;
	d_vec_hullSet = *hullSet;
}

void SetEllipse(std::vector<ellipse> *ellipseSet)
{
	if(ellipseSet->size() > MAX_NUM_GROUP)
	{
		cout<<"exceeded the maximum number of groups..."<<endl;
		exit(1);
	}
	//_ellipseSet = ellipseSet;
	_h_vec_ellipseSet = ellipseSet;
}

void SetPickedLineSet(thrust::host_vector<int> *pickedLineSet)
{
	_pickedLineSet = pickedLineSet;
}

//void UpdateLineIndexFromOffsetLengths()
//{
//
//}

void SetConnectivity(thrust::host_vector<int> &length, thrust::host_vector<int> &offset)
{
	int numLines = length.size();
	d_vec_streamlineLengths.resize(numLines);
	d_vec_streamlineOffsets.resize(numLines);
	thrust::copy(length.begin(), length.end(), d_vec_streamlineLengths.begin());
	//d_vec_streamlineLengthsOrig = d_vec_streamlineLengths;
	//d_vec_streamlineOffsetsOrig = d_vec_streamlineOffsets;
	//cout<<"size of d_vec_streamlineLengths:"<<d_vec_streamlineLengths.size();
	thrust::exclusive_scan(d_vec_streamlineLengths.begin(), d_vec_streamlineLengths.end(), d_vec_streamlineOffsets.begin());
	//cout<<"size of offsets:"<<d_vec_streamlineOffsets.end() - d_vec_streamlineOffsets.begin()<<endl;
	offset = d_vec_streamlineOffsets;
	//for(int i = 0; i < h_vec_offset.size(); i++)
	//	offset.push_back(h_vec_offset[i]);
	d_vec_streamlineLengthsOrig = d_vec_streamlineLengths;
	d_vec_streamlineOffsetsOrig = d_vec_streamlineOffsets;
	
	//thrust::for_each(
	//	thrust::make_zip_iterator(thrust::make_tuple(d_vec_streamlineOffsets.begin(), d_vec_streamlineLengths.begin(), counting_zero)),
	//	thrust::make_zip_iterator(thrust::make_tuple(d_vec_streamlineOffsets.end(), d_vec_streamlineLengths.end(), counting_zero + d_vec_streamlineOffsets.size())),
	//	functor_GenLineIndex(thrust::raw_pointer_cast(_d_vec_lineIndexOrig.data())));
	UpdateLineIndex();
	_d_vec_lineIndexOrig = _d_vec_lineIndex;

	thrust::for_each(
		d_vec_streamlineOffsets.begin(),
		d_vec_streamlineOffsets.end(),
	functor_UpdateCutPointsMarkByConnectivity(
		thrust::raw_pointer_cast(_d_vec_cutPointsMarkOrig.data())
		));
	_d_vec_cutPointsMark = _d_vec_cutPointsMarkOrig;


	check_cuda_errors(__FILE__, __LINE__);
}

void GetConnectivity(thrust::host_vector<int> &offsets, thrust::host_vector<int> &lengths)
{
	lengths = d_vec_streamlineLengths;
	offsets = d_vec_streamlineOffsets;
}

void SetLineIndexCUDA(int *data)
{
	_d_vec_lineIndex.assign(data, data + _nv);
	//cout<<"_nv line index:" << _nv <<endl;
}

void SetDeformOnPara(bool *deformOn)
{
	 _deformOn = deformOn;
}

void AssignLineIndexFromDevice(int *data)
{
	//cout<<"AssignLineIndexFromDevice..."<<endl;
	thrust::copy(_d_vec_lineIndex.begin(), _d_vec_lineIndex.end(), data);
}

void SetParaCUDA(float para)
{
	_para = para;
}

void SetVBOData(float4* d_raw_clip, float3* d_raw_tangent, int* d_raw_translucent)
{
	_d_ptr_posClip = thrust::device_pointer_cast(d_raw_clip);
	_d_ptr_tangent = thrust::device_pointer_cast(d_raw_tangent);
	_d_ptr_translucent = thrust::device_pointer_cast(d_raw_translucent);
}

void SetDeformWinSize(int w, int h)
{
	hipMemcpyToSymbol(HIP_SYMBOL(WIN_WIDTH), &w, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(WIN_HEIGHT), &h, sizeof(int));
	_h_winWidth = w;
	_h_winHeight = h;
}

void SetMatrix(	float* modelview, float* projection, float* invModelView, float* invProjection)
{
	hipMemcpyToSymbol(HIP_SYMBOL(MODELVIEW), modelview, 16 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(INVMODELVIEW), invModelView, 16 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(PROJECTION), projection, 16 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(INVPROJECTION), invProjection, 16 * sizeof(float));

	for(int i = 0; i < 16; i++)
	{
		_h_modelview[i] = modelview[i];
		_h_projection[i] = projection[i];
	}
	_invModelView = invModelView;
	_invProjection = invProjection;

}

thrust::host_vector<float2> GetPosScreenOrig()
{
	thrust::device_vector<float2> d_vec_origPosScreen(_nv);
	thrust::device_vector<float4> d_vec_origPosClip(_nv);
	thrust::transform(_d_vec_origPos.begin(), _d_vec_origPos.end(), d_vec_origPosClip.begin(), functor_Object2Clip());
	thrust::transform(d_vec_origPosClip.begin(), d_vec_origPosClip.end(), d_vec_origPosScreen.begin(), functor_Clip2Screen());

	thrust::host_vector<float2> h_vec_screen  = d_vec_origPosScreen;
	return h_vec_screen;
}


	//thrust::for_each(
	//	thrust::make_zip_iterator(thrust::make_tuple(d_vec_posScreen.begin(), _d_vec_vertexIsFocus.begin(), _d_ptr_translucent)),
	//	thrust::make_zip_iterator(thrust::make_tuple(d_vec_posScreen.end(), _d_vec_vertexIsFocus.end(), _d_ptr_translucent + _nv)),
	//	functor_UpdateTranslucentVertices(
	//		thrust::raw_pointer_cast(d_vec_ellipseSet.data()),
	//		_h_vec_ellipseSet->size())
	//		);

struct functor_UpdateTranslucentVertices
{
	ellipse* ellipseSet;
	int numEllipses;

	template <typename T>
	__device__ void operator() (T t)
	{
		float2 posScreen = thrust::get<0>(t);
		bool vertexIsFocus = thrust::get<1>(t);
		bool insideAnyBlades = false;
		for(int j = 0; j < numEllipses; j++)
		{
			//ellipse e = ellipseSet[j];
			//float2 line[2];
			//line[0] = make_float2(e.x - e.a * cos(e.angle), e.y - e.a * sin(e.angle));
			//line[1] = make_float2(e.x + e.a * cos(e.angle), e.y + e.a * sin(e.angle));
			//float2 center = ProjectPoint2Line(posScreen, line[0], line[1]);
			//float radius = radius_blade(ellipseSet[j], center);
		//	if(length(posScreen - tmp) < ellipseSet[j].a)//inOneBlade(ellipseSet[j], 0.5/*ellipseSet[j].b*/, posScreen))
		//	if(length(center - posScreen) < radius)
			if(inOneEllipse(ellipseSet[j], posScreen))
				insideAnyBlades = true;
		}
		if(insideAnyBlades && !vertexIsFocus)
			thrust::get<2>(t) = 1;
		else
			thrust::get<2>(t) = 0;
	}

	functor_UpdateTranslucentVertices(
		ellipse* _ellipseSet, int _numEllipses)
	{
		ellipseSet = _ellipseSet;
		numEllipses = _numEllipses;
	}
};


// Wrapper for the __global__ call that sets up the kernel call
void launch_kernel(clock_t t0)//, unsigned int mesh_width, unsigned int mesh_height, float time)
{
//	clock_t t0;
//#if (TEST_PERFORMANCE == 2)
////	t0 = clock();
//#endif
	_d_vec_prePos = _d_vec_pos;
	float4* d_raw_ptr_pos = thrust::raw_pointer_cast(_d_vec_pos.data());

    // execute the kernel
    dim3 block(256, 1, 1);
    dim3 grid(ceil((float)_nv / block.x), 1, 1);

	thrust::for_each(
		thrust::make_zip_iterator(thrust::make_tuple(counting_zero, _d_ptr_tangent)),
		thrust::make_zip_iterator(thrust::make_tuple(counting_zero + _nv, _d_ptr_tangent + _nv)),
		functor_computeNormal(thrust::raw_pointer_cast(_d_vec_lineIndex.data()), d_raw_ptr_pos));

	//clip coordiates of streamlines
	thrust::transform(_d_vec_pos.begin(), _d_vec_pos.end(), _d_ptr_posClip, functor_Object2Clip());
	thrust::transform(_d_ptr_posClip, _d_ptr_posClip + _nv, d_vec_posScreen.begin(), functor_Clip2Screen());

	if(0 == _h_vec_ellipseSet->size() )
		return;

	thrust::device_vector<ellipse> d_vec_ellipseSet = *_h_vec_ellipseSet;
	//cout<<"d_vec_ellipseSet.size():"<<d_vec_ellipseSet.size()<<endl;
	thrust::for_each(
		thrust::make_zip_iterator(thrust::make_tuple(d_vec_posScreen.begin(), _d_vec_vertexIsFocus.begin(), _d_ptr_translucent)),
		thrust::make_zip_iterator(thrust::make_tuple(d_vec_posScreen.end(), _d_vec_vertexIsFocus.end(), _d_ptr_translucent + _nv)),
		functor_UpdateTranslucentVertices(
			thrust::raw_pointer_cast(d_vec_ellipseSet.data()),
			d_vec_ellipseSet.size())
			);

#if (TEST_PERFORMANCE == 2)
	PrintElapsedTime(t0, "prepare data(before deformation kernel)");
#endif	
	if(*_deformOn)
	{
#if (TEST_PERFORMANCE == 3)
	    hipEvent_t start, stop;
		float time;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipEventRecord(start, 0);
#endif
		//cout<<"_d_vec_lineIndex.back() before kernel:"<<_d_vec_lineIndex.back()<<endl;
   		kernel_convex<<< grid, block>>>(d_raw_ptr_pos, thrust::raw_pointer_cast(_d_ptr_posClip), 
					thrust::raw_pointer_cast(d_vec_posScreen.data()), thrust::raw_pointer_cast(_d_vec_prePos.data()), 
					thrust::raw_pointer_cast(_d_vec_origPos.data()), thrust::raw_pointer_cast(_d_vec_lineIndex.data()),
					thrust::raw_pointer_cast(_d_vec_vertexIsFocus.data()),
					_nv,
					thrust::raw_pointer_cast(d_vec_ellipseSet.data()), 
					thrust::raw_pointer_cast(d_vec_hullSet.data()), 
					_h_vec_ellipseSet->size(),
					*_deformMode,
					_para);
		 
#if (TEST_PERFORMANCE == 3)
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);

		hipEventElapsedTime(&time, start, stop);
		printf("%f\tms to %s\n", time, "run deformation kernel");
#endif
		check_cuda_errors(__FILE__, __LINE__);
	}
}
